
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void MatMulKernel(double *A, double *B, double *C, int ACBR, int AR, int BC){
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = .0;
    if (r < BC && c < AR){
        for(int i = 0; i<ACBR; i++){
            sum += A[r*ACBR+i]*B[i*ACBR+c];
        }
    }
    C[r*ACBR+c]=sum;
};

//# of matrix A columns === # of matrix b rows 
//ACBR (A Col B Row)
//AR A Row
//BC B Col

double **MatMul(double **A, double **B, int ACBR, int AR, int BC){

    double **Host_A = (double**)malloc(AR*sizeof(double*));
    double *Contig_Block_A = (double*) malloc(AR * ACBR * sizeof(double));
    for (int i = 0; i<AR; i++){
        Host_A[i] = &Contig_Block_A[i*ACBR];
    }

    double **Host_B = (double**)malloc(ACBR*sizeof(double*));
    double *Contig_Block_B = (double*)malloc(ACBR * BC *sizeof(double));
    for(int i = 0; i<ACBR; i++){
        Host_B[i] = &Contig_Block_B[i*BC];
    }

    double **Host_Result = (double**)malloc(AR*sizeof(double*));
    double *Contig_Block_Result = (double*)malloc(AR * BC *sizeof(double));
    for(int i = 0; i<AR; i++){
        Host_Result[i] = &Contig_Block_Result[i*BC];
    }

    double *GPU_A, *GPU_B, *GPU_Result;

    hipMalloc(&GPU_A,AR*ACBR);
    hipMalloc(&GPU_B,ACBR*BC);
    hipMalloc(&GPU_Result,AR*BC);

    hipMemcpy(GPU_A,Host_A,AR,hipMemcpyHostToDevice);
    hipMemcpy(GPU_B,Host_B,ACBR,hipMemcpyHostToDevice);
    hipMemcpy(GPU_Result,Host_Result,AR,hipMemcpyHostToDevice);

    dim3 dimBlock(AR);
    dim3 dimGrid(1,1);

    MatMulKernel<<<dimGrid,dimBlock>>>(GPU_A,GPU_B,GPU_Result,ACBR,AR,BC);

    hipMemcpy(GPU_Result,Host_Result,AR*BC,hipMemcpyDeviceToHost);

    hipFree(GPU_A);
    hipFree(GPU_B);
    hipFree(GPU_Result);

    free(Host_A);
    free(Contig_Block_A);
    free(Host_B);
    free(Contig_Block_B);

    return Host_Result;
}

